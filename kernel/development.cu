#include "hip/hip_runtime.h"
#include "development.cuh"

#include "cuda_utils.cuh"
#include "environment.h"
#include "phenotype_program.h"

namespace epigenetic_gol_kernel {
namespace {

// Special value for coordinate transforms, indicating that a Cell is no longer
// part of the logical space of the board and should not receive any value.
const int OUT_OF_BOUNDS = -1;

__device__ Scalar get_scalar(
        const ScalarArgument& arg, const Genotype& genotype) {
    switch (arg.bias_mode) {
        case BiasMode::FIXED_VALUE:
            return arg.bias;
        default:
            return genotype.scalar_genes[arg.gene_index % NUM_GENES];
    }
}

__device__ const Stamp& get_stamp(
        const StampArgument& arg, const Genotype& genotype) {
    switch (arg.bias_mode) {
        case BiasMode::FIXED_VALUE:
            return arg.bias;
        default:
            return genotype.stamp_genes[arg.gene_index % NUM_GENES];
    }
}

__device__ void apply_array_1d(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int row_offset = get_scalar(op.args[0], genotype) % WORLD_SIZE;
    int col_offset = get_scalar(op.args[1], genotype) % WORLD_SIZE;

    int rep_number = min(row / row_offset, col / col_offset);
    row -= rep_number * row_offset;
    col -= rep_number * col_offset;
}

__device__ void apply_array_2d(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int row_offset = get_scalar(op.args[0], genotype) % WORLD_SIZE;
    int col_offset = get_scalar(op.args[1], genotype) % WORLD_SIZE;

    row %= row_offset;
    col %= col_offset;
}

__device__ void apply_copy(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int row_offset = get_scalar(op.args[0], genotype) % WORLD_SIZE;
    int col_offset = get_scalar(op.args[1], genotype) % WORLD_SIZE;

    int rep_number = min(min(row / row_offset, col / col_offset), 1);
    row -= rep_number * row_offset;
    col -= rep_number * col_offset;
}

template<int SIZE>
__device__ void apply_crop(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int row_offset = get_scalar(op.args[0], genotype) % SIZE;
    int col_offset = get_scalar(op.args[1], genotype) % SIZE;
    row = (row < row_offset) ? row : OUT_OF_BOUNDS;
    col = (col < col_offset) ? col : OUT_OF_BOUNDS;
}

template<int SIZE>
__device__ void apply_flip(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int axes = get_scalar(op.args[0], genotype);
    row = (axes & 0b01) ? SIZE - 1 - row : row;
    col = (axes & 0b10) ? SIZE - 1 - col : col;
}

template<int SIZE>
__device__ void apply_mirror(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int axes = get_scalar(op.args[0], genotype);
    row = (axes & 0b01) && row >= SIZE / 2 ? SIZE - 1 - row : row;
    col = (axes & 0b10) && col >= SIZE / 2 ? SIZE - 1 - col : col;
}

template<int SIZE>
__device__ void apply_quarter(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int axes = get_scalar(op.args[0], genotype);
    constexpr int half = SIZE / 2;
    unsigned char quadrant_bitmask = 1 << ((row >= half) | (col >= half) << 1);
    row = axes & quadrant_bitmask ? row : OUT_OF_BOUNDS;
    col = axes & quadrant_bitmask ? col : OUT_OF_BOUNDS;
}

template<int SIZE>
__device__ void apply_rotate(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int rotation = get_scalar(op.args[0], genotype) % 4;
    if (rotation == 0) {
        return;
    } else if (rotation == 1) {
        int old_row = row;
        row = SIZE - 1 - col;
        col = old_row;
    } else if (rotation == 2) {
        row = SIZE - 1 - row;
        col = SIZE - 1 - col;
    } else if (rotation == 3) {
        int old_row = row;
        row = col;
        col = SIZE - 1 - old_row;
    }
}

__device__ void apply_scale(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    constexpr int max_scale = STAMP_SIZE;
    int row_scale = (get_scalar(op.args[0], genotype) - 1) % max_scale + 1;
    int col_scale = (get_scalar(op.args[1], genotype) - 1) % max_scale + 1;
    row = row / row_scale;
    col = col / col_scale;
}

__device__ void apply_tile(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int offset = get_scalar(op.args[0], genotype) % STAMP_SIZE;
    bool flip_every_other = get_scalar(op.args[1], genotype) & 0b1;

    bool every_other_row = (row / STAMP_SIZE) & 0b1;
    offset = every_other_row ? offset : 0;
    bool every_other_col = ((col + offset) / STAMP_SIZE) & 0b1;
    bool flip = flip_every_other && (every_other_row ^ every_other_col);

    row = flip ? STAMP_SIZE - 1 - row % STAMP_SIZE : row % STAMP_SIZE;
    col = flip ? (STAMP_SIZE - 1 - col + offset) % STAMP_SIZE
               : (col + offset) % STAMP_SIZE;
}

template<int SIZE>
__device__ void apply_translate(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    int row_offset = get_scalar(op.args[0], genotype) % SIZE;
    int col_offset = get_scalar(op.args[1], genotype) % SIZE;
    row = (row - row_offset) % SIZE;
    col = (col - col_offset) % SIZE;
}

template<int SIZE>
__device__ void apply_transform(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col);

// TODO: Switch-based dispatching is kinda slow. It would be better to use
// vtable or tag-based based dispatching, but those don't apply when the
// types aren't known in advance. It would likely require a "compilation"
// pass over the program, to resolve they types once up front and embodying the
// results in a data structure or dynamically generated PTX file. It might be
// more elegant to work with the Python data structure directly rather than
// using these structs as an intermediate structure, but that may be awkward if
// using vtable lookup, since the compiled objects would have to be built on
// the GPU for that to work, which means passing Python objects to the GPU.
template<>
__device__ void apply_transform<WORLD_SIZE>(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    // Apply global transformations. Note that some transformations can be used
    // either globally or locally. They get configured by a template argument.
    switch (op.type) {
        case TransformType::ARRAY_1D:
            return apply_array_1d(genotype, op, row, col);
        case TransformType::ARRAY_2D:
            return apply_array_2d(genotype, op, row, col);
        case TransformType::COPY:
            return apply_copy(genotype, op, row, col);
        case TransformType::CROP:
            return apply_crop<WORLD_SIZE>(genotype, op, row, col);
        case TransformType::FLIP:
            return apply_flip<WORLD_SIZE>(genotype, op, row, col);
        case TransformType::MIRROR:
            return apply_mirror<WORLD_SIZE>(genotype, op, row, col);
        case TransformType::QUARTER:
            return apply_quarter<WORLD_SIZE>(genotype, op, row, col);
        case TransformType::ROTATE:
            return apply_rotate<WORLD_SIZE>(genotype, op, row, col);
        case TransformType::SCALE:
            return apply_scale(genotype, op, row, col);
        case TransformType::TILE:
            return apply_tile(genotype, op, row, col);
        case TransformType::TRANSLATE:
            return apply_translate<WORLD_SIZE>(genotype, op, row, col);
        default:
            return;
    }
}

template<>
__device__ void apply_transform<STAMP_SIZE>(
        const Genotype& genotype, const TransformOperation& op,
        int& row, int& col) {
    // Only some of the transform operations actually make sense to apply to
    // the stamp coordinate space. Any others will be ignored.
    switch (op.type) {
        case TransformType::CROP:
            return apply_crop<STAMP_SIZE>(genotype, op, row, col);
        case TransformType::FLIP:
            return apply_flip<STAMP_SIZE>(genotype, op, row, col);
        case TransformType::MIRROR:
            return apply_mirror<STAMP_SIZE>(genotype, op, row, col);
        case TransformType::QUARTER:
            return apply_quarter<STAMP_SIZE>(genotype, op, row, col);
        case TransformType::ROTATE:
            return apply_rotate<STAMP_SIZE>(genotype, op, row, col);
        case TransformType::SCALE:
            return apply_scale(genotype, op, row, col);
        case TransformType::TRANSLATE:
            return apply_translate<STAMP_SIZE>(genotype, op, row, col);
        default:
            return;
    }
}

template<int SIZE>
__device__ void apply_transform_list(
        const Genotype& genotype, const TransformOperation* transforms,
        int& row, int& col) {
    // Go through the array of transforms, applying them to row and col to
    // remap the coordinate space we're drawing on.
    for (int i = 0; i < MAX_TRANSFORMS; i++) {
        // A NONE transform has no effect and indicates the end of this part of
        // the program. Stop processing the transform list.
        if (transforms[i].type == TransformType::NONE) {
            break;
        }
        // If this position has been marked out of bounds by a previous
        // transform (like a CROP operation), then no further transforms get to
        // apply here.
        if (row == OUT_OF_BOUNDS || col == OUT_OF_BOUNDS) {
            break;
        }
        // Otherwise, keep transforming row and col.
        apply_transform<SIZE>(genotype, transforms[i], row, col);
    }
}

__device__ bool apply_draw(
        const Genotype& genotype, const DrawOperation& draw_op,
        int& row, int& col) {
    // Transform the global coordinate space. This makes it possible to
    // position the stamp anywhere, repeat the stamp, and warp it in various
    // ways. Originally, row and col indicate the distance from the top-left
    // corner, but once all the transforms are applied, they indicate where in
    // the stamp to draw from.
    apply_transform_list<WORLD_SIZE>(
            genotype, draw_op.global_transforms, row, col);

    // Make sure not to read data from beyond the extents of the stamp found in
    // the genotype. Anything out of bounds for the stamp will be empty space.
    // This is computed BEFORE applying the stamp transforms so they won't
    // distort the result by modifying row and col further.
    bool in_bounds = (row >= 0 && row < STAMP_SIZE &&
                      col >= 0 && col < STAMP_SIZE);

    // Transform the stamp coordinate space. This allows for neutral mutations
    // to the data being drawn.
    apply_transform_list<STAMP_SIZE>(
            genotype, draw_op.stamp_transforms, row, col);

    // Update the in_bounds calculation after the additional transforms.
    in_bounds &= (row >= 0 && row < STAMP_SIZE &&
                  col >= 0 && col < STAMP_SIZE);

    // Actually fetch the relevant data from the genotype and determine whether
    // this cell should be set alive or not.
    const Stamp& stamp = get_stamp(draw_op.stamp, genotype);
    return in_bounds && stamp[row][col] == Cell::ALIVE;
}

} // namespace

__device__ void make_phenotype(
        const PhenotypeProgram& program, const Genotype& genotype,
        const int& row, const int& col, Cell& cell) {
    // The first draw operation always applies directly to the phenotype. After
    // that, additional draw operations layer over what came before using
    // whatever composition is specified.
    bool alive = false;
    for (int i = 0; i < MAX_DRAWS; i++) {
        if (program.draw_ops[i].compose_mode == ComposeMode::NONE) {
            break;
        }
        // Make a copy of row / col for this draw operation. This way, each
        // draw operation can transform the coordinate space independently.
        int r = row;
        int c = col;
        // Figure out whether this draw operation wants to set the cell to dead
        // or alive, then merge that with the previous values.
        const bool new_value = apply_draw(genotype, program.draw_ops[i], r, c);
        switch (program.draw_ops[i].compose_mode) {
            case ComposeMode::OR:
                alive |= new_value;
                break;
            case ComposeMode::XOR:
                alive ^= new_value;
                break;
            case ComposeMode::AND:
                alive &= new_value;
                break;
            default:
                break;
        };
    }
    // Actually modify the cell value based on composing all the draw
    // operations. Note, the code above uses bool values only to translate into
    // Cell values here because treating ALIVE and DEAD as bools has
    // counter-intuitive behavior. The value of ALIVE is 0x00 so that a live
    // cell appears black in the output images.
    // TODO: Would it be better to flip this, let DEAD be 0x00 and perform an
    // inversion when rendering the phenotypes and simulation videos?
    cell = alive ? Cell::ALIVE : Cell::DEAD;
}

} // namespace epigenetic_gol_kernel

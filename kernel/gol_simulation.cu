#include "hip/hip_runtime.h"
#include "gol_simulation.h"

#include <hipcub/hipcub.hpp>

#include "cuda_utils.cuh"
#include "development.cuh"
#include "fitness.cuh"

namespace epigenetic_gol_kernel {
namespace {

constexpr int CELLS_PER_THREAD = 8;
constexpr int REPEATS_PER_ROW = WORLD_SIZE / CELLS_PER_THREAD;
constexpr int THREADS_PER_BLOCK = WORLD_SIZE * REPEATS_PER_ROW;

__device__ __host__ Cell get_next_state(
        const int& curr_row, const int& curr_col, const Frame& last_frame) {
    // Count up neighbors of this Cell that are ALIVE by looking at all the
    // adjacent Cells that are in bounds for this Frame. Bounds checking is
    // done with min / max which is faster than using ifs or ternaries.
    const int prev_row = max(curr_row - 1, 0);
    const int next_row = min(curr_row + 1, WORLD_SIZE - 1);
    const int prev_col = max(curr_col - 1, 0);
    const int next_col = min(curr_col + 1, WORLD_SIZE - 1);
    const int neighbors = (
            (last_frame[prev_row][prev_col] == Cell::ALIVE) +
            (last_frame[prev_row][curr_col] == Cell::ALIVE) +
            (last_frame[prev_row][next_col] == Cell::ALIVE) +
            (last_frame[curr_row][prev_col] == Cell::ALIVE) +
            (last_frame[curr_row][next_col] == Cell::ALIVE) +
            (last_frame[next_row][prev_col] == Cell::ALIVE) +
            (last_frame[next_row][curr_col] == Cell::ALIVE) +
            (last_frame[next_row][next_col] == Cell::ALIVE));

    // Compute the next state for this Cell from the previous state and the
    // number of living neighbors.
    const Cell& last_state = last_frame[curr_row][curr_col];
    return (last_state == Cell::ALIVE && (neighbors == 2 || neighbors == 3) ||
            last_state == Cell::DEAD && neighbors == 3)
        ? Cell::ALIVE : Cell::DEAD;
}

__global__ void GolKernel(
        const FitnessGoal goal,
        const PhenotypeProgram* programs,
        const Genotype* genotypes,
        Video* videos,
        Fitness* fitness_scores,
        bool record) {
    const int& species_index = blockIdx.y;
    const int population_index = blockIdx.y * gridDim.x + blockIdx.x;
    const int row = threadIdx.x / REPEATS_PER_ROW;
    const int col = CELLS_PER_THREAD * (threadIdx.x % REPEATS_PER_ROW);

    const PhenotypeProgram& program = programs[species_index];
    const Genotype& genotype = genotypes[population_index];
    Video& video = videos[population_index];
    Fitness& fitness = fitness_scores[population_index];

    // Calculating the next frame of a Game of Life simulation requires looking
    // at all of a Cell's neighbors in the previous frame. Since it takes many
    // warps to compute a single Frame, this means adjacent warps need to share
    // state for the previous frame (but not the next one). We store one whole
    // Frame in shared memory (in theory you only need enough to represent the
    // seams between warps, but that scheme would require extra work) and just
    // the cells for the next frame being computed by this thread in registers.
    __shared__ Frame last_frame;
    Cell curr_frame[CELLS_PER_THREAD];

    // Each frame may contribute to overall fitness. This function doesn't
    // always record all those frames, so we compute fitness incrementally
    // frame by frame, storing partial work here.
    PartialFitness partial_fitness[CELLS_PER_THREAD];

    // Interpret this organism's genotype to generate the phenotype, which is
    // the first frame of the simulation.
    for (int i = 0; i < CELLS_PER_THREAD; i++) {
        // TODO: Pass last_frame as a workspace for the stack? It only has
        // depth one, but that's enough for most simple compositions.
        make_phenotype(program, genotype, row, col+i, curr_frame[i]);
    }

    // Make sure this frame is finished before looking at it.
    __syncthreads();

    // Run the simulated lifetime...
    for (int step = 0; step < NUM_STEPS; step++) {
        // Copy the most recently computed frame data into shared memory and
        // wait for it to finish before calling get_next_state below. Since
        // each thread works on CELLS_PER_THREAD contiguous Cells, we can do
        // this with a single memcpy instead of a loop. Doing this well before
        // it's needed seems to help hide the memory access latency.
        memcpy(&last_frame[row][col], curr_frame, sizeof(curr_frame));
        __syncthreads();

        // If recording, save a copy of each frame to global memory.
        if (record) {
            memcpy(&video[step][row][col], curr_frame, sizeof(curr_frame));
        }

        // Compute the fitness contribution of each frame as we go along.
        for (int i = 0; i < CELLS_PER_THREAD; i++) {
            update_fitness(
                    step, row, col+i, curr_frame[i], goal, partial_fitness[i]);
        }

        // If we've already computed, evaluated, and saved the last frame, then
        // stop here before computing another one.
        if (step == NUM_STEPS - 1) break;

        // Compute the next frame from the previous one.
        for (int i = 0; i < CELLS_PER_THREAD; i++) {
            curr_frame[i] = get_next_state(row, col+i, last_frame);
        }
        __syncthreads();
    }

    // Finalize all the fitness scores.
    for (int i = 0; i < CELLS_PER_THREAD; i++) {
        finalize_fitness(goal, partial_fitness[i]);
    }

    // Sum fitness contributions from each thread.
    auto reduce = hipcub::BlockReduce<Fitness, THREADS_PER_BLOCK>();
    int sum = reduce.Sum((Fitness(&)[CELLS_PER_THREAD]) partial_fitness);

    // Save the final result to global memory to return to the host.
    if (threadIdx.x == 0) {
        fitness = sum;
    }
}

} // namespace

void simulate_population(
        const unsigned int population_size,
        const unsigned int num_species,
        const FitnessGoal& goal,
        const PhenotypeProgram* programs,
        const Genotype* genotypes,
        Video* videos,
        Fitness* fitness_scores,
        bool record) {
    GolKernel<<<
        { population_size / num_species, num_species },
        THREADS_PER_BLOCK
    >>>(goal, programs, genotypes, videos, fitness_scores, record);
    CUDA_CHECK_ERROR();
}

Video* simulate_phenotype(const Frame& phenotype) {
    Video* video = (Video*) new Video;
    // Fill in the first frame of the Video from the phenotype
    memcpy(video, &phenotype, sizeof(Frame));
    // Compute the remaining frames from the first one.
    for (int step = 1; step < NUM_STEPS; step++) {
        for (int row = 0; row < WORLD_SIZE; row++) {
            for (int col = 0; col < WORLD_SIZE; col++) {
                (*video)[step][row][col] = 
                    get_next_state(row, col, (*video)[step-1]);
            }
        }
    }
    return video;
}

namespace {

__global__ void MakePhenotypeKernel(
        const PhenotypeProgram& program,
        const Genotype* genotypes,
        Frame* phenotypes) {
    const int population_index = blockIdx.x;
    const int row = threadIdx.x / REPEATS_PER_ROW;
    const int col = CELLS_PER_THREAD * (threadIdx.x % REPEATS_PER_ROW);

    const Genotype& genotype = genotypes[population_index];
    Frame& phenotype = phenotypes[population_index];

    for (int i = 0; i < CELLS_PER_THREAD; i++) {
        make_phenotype(program, genotype, row, col+i,
                (Cell&) phenotype[row][col+i]);
    }
}

} // namespace

const Frame* render_phenotype(
        const PhenotypeProgram& h_program,
        const Genotype* h_genotype) {
    DeviceData<PhenotypeProgram> program(&h_program);
    DeviceData<Genotype> genotype;
    // Either use the given Genotype, or use an empty one as a default.
    if (h_genotype) {
        genotype.copy_from_host(h_genotype);
    } else {
        CUDA_CALL(hipMemset(genotype, 0, sizeof(Genotype)));
    }
    DeviceData<Frame> phenotype;

    MakePhenotypeKernel<<<
        1, THREADS_PER_BLOCK
    >>>(program, genotype, phenotype);
    return phenotype.copy_to_host();
}
} // namespace epigenetic_gol_kernel
